#include "hip/hip_runtime.h"
/*  

    3D Cellular Automata Simulation,
    Runs on CUDA
    - Sparsh

    OpenGL code referenced from from: https://docs.nvidia.com/cuda/cuda-samples/index.html#simple-opengl

*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>


#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1024;
const unsigned int window_height = 1024;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;
const unsigned int mesh_length = 256;


// vbo variables
GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;
void* d_vbo_buffer = NULL;

GLuint loc_vbo;
struct hipGraphicsResource* cuda_loc_vbo_resource;
void* d_loc_vbo_buffer = NULL;

int* state;
int* state_next;

int* d_state;
int* d_state_next;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int* pArgc = NULL;
char** pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char** argv, char* ref_file);
void cleanup();

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res,
    unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);


// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource, int* state, int* state_next, int* d_state, int* d_state_next);
void runAutoTest(int devID, char** argv, char* ref_file);
void checkResultCuda(int argc, char** argv, const GLuint& vbo);

const char* sSDKsample = "simpleGL (VBO)";

/*
    starting config of the state, input is array of coords you want to spawn live cell
*/
void initializeStartState(int x[], int y[], int z[], int count) {


    // casual fallback on null input, TODO refactor to remove this
    if (x == NULL) {


        for(int x = 0; x < mesh_width; x++)
            for (int y = 0; y < mesh_width; y++)
                for (int z = 0; z < mesh_width; z++)
                {
                    int loc = (mesh_width * (y * mesh_width + x)) + z;
                    state[loc] = 0;
                    state_next[loc] = 0;
                }


        int mid = mesh_width / 2;
        int tx, ty, tz;
        tx = mid;
        ty = mid;
        tz = mid;

        int loc = (mesh_width * (ty * mesh_width + tx)) + tz;


        state[loc] = 1;
       

        tx = mid + 1;
        ty = mid + 1;
        tz = mid + 1;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;


        tx = mid - 1;
        ty = mid - 1;
        tz = mid - 1;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;
        


        /*tx = mid + 1;
        ty = mid;
        tz = mid + 1;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;*/

        /*tx = mid;
        ty = mid + 1;
        tz = mid;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;*/

        

        /*tx = mid + 8;
        ty = mid - 8;
        tz = mid - 8;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;
        
        tx = mid + 8;
        ty = mid - 8;
        tz = mid - 7;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;

        tx = mid + 8;
        ty = mid - 7;
        tz = mid - 8;
        loc = (mesh_width * (ty * mesh_width + tx)) + tz;

        state[loc] = 1;
        */


        return;
    }

    for (int i = 0; i < count; i++) {
        int loc = (mesh_width * (y[i] * mesh_width + x[i])) + z[i];
        state[loc] = 1;
    }

}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int length, unsigned int width, unsigned int height, float time, int* state, int* state_next)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    int my_offset = (width * (y * width + x)) + z;
    
    //printf("\nI am %d, %d, %d as %d", x,y,z,my_offset);

    // calculate uvw coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    float w = z / (float)length;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;
    w = w * 2.0f - 1.0f;

    //evolution rules
    int count_live = 0, count_dead = 0;
    
    //thank you ms excel
    int dx[] = {-1, -1, -1, -1, -1, -1, -1, -1, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    int dy[] = {-1, -1, -1, 0, 0, 0, 1, 1, 1, -1, -1, -1, 0, 0, 0, 1, 1, 1, -1, -1, -1, 0, 0, 0, 1, 1, 1};
    int dz[] = {-1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1};

    for (int i = 0; i < 27; i++) {
       
       
        int nx = x + dx[i], ny = y + dy[i], nz = z + dz[i];
        
        if (MAX(MAX(nx, ny), nz) >= width || MIN(MIN(nx, ny), nz) < 0) {
            continue;
        }

        if (nx == x && ny == y && nz == z) {
            continue;
        }
            
        int offset = (width * (ny * width + nx)) + nz;

        if (state[offset] == 1) {
            count_live++;
            //printf("\nvisiting xyz = %d %d %d  neighbor = %d %d %d found alive cell alive count is %d", x, y, z, nx, ny, nz, count_live);
        }else count_dead++;
    
    }

    //basic test rule, spawn new if neighborhood has 2 alive and im empty
    // & write output vertex
    if (state[my_offset] == 0 && count_live == 2) {
        
        //printf("\noffset in cuda: %d, xyz %d %d %d, uvw %f %f %f", my_offset, x, y, z, u, v, w);
        state_next[my_offset] = 1;
        pos[my_offset] = make_float4(u, v, w, 1.0f);
    }
    else if (state[my_offset] == 1 && count_live >= 6) {

        state_next[my_offset] = 0;
        pos[my_offset] = make_float4(1, 1, 1, 1.0f);
    }
    else {
        //printf("\nelse offset in cuda: %d, xyz %d %d %d, uvw %f %f %f", my_offset, x, y, z, u, v, w);
        state_next[my_offset] = state[my_offset];
        pos[my_offset] = make_float4(1, 1, 1, 1.0f);
    }

}


void launch_kernel(float4* pos, unsigned int mesh_width,
    unsigned int mesh_height, float time, int* d_state, int* d_state_next)
{
    // execute the kernel 
    dim3 block(8, 8, 8); 
    dim3 grid(mesh_width / block.x, mesh_width / block.y, mesh_height / block.z);
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_width, mesh_height, time, d_state, d_state_next);
}

bool checkHW(char* name, const char* gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    char* ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif

    printf("%s starting...@main1 !!!\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char**)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char**)argv, "file", (char**)&ref_file);
        }
    }

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

    // initialize necessary OpenGL extensions
    if (!isGLVersionSupported(2, 0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char** argv, char* ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    // command line mode only
    if (ref_file != NULL)
    {
        // create VBO
        checkCudaErrors(hipMalloc((void**)&d_vbo_buffer, mesh_width * mesh_height * 4 * sizeof(float)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // register callbacks
        glutDisplayFunc(display);  
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);

        glutCloseFunc(cleanup);

        // create VBO
        createVBO(&loc_vbo, &cuda_loc_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
  
        //allocate state grid on host
        state = (int*)malloc(mesh_width * mesh_width * mesh_height * sizeof(int));
        state_next = (int*)malloc(mesh_width * mesh_width * mesh_height * sizeof(int));

        //initialize first state
        initializeStartState(NULL, NULL, NULL, NULL);


        // run the cuda part
        runCuda(&cuda_loc_vbo_resource, state, state_next, d_state, d_state_next);
        
        // start rendering mainloop
        glutMainLoop();
    }

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource, int* state, int* state_next, int* d_state, int* d_state_next)
{
    // map OpenGL buffer object for writing from CUDA
    float4* dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes,
        *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    //allocate device memory
    checkCudaErrors(hipMalloc((void**)&d_state, mesh_width * mesh_width * mesh_height * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_state_next, mesh_width * mesh_width * mesh_height * sizeof(int)));
    //printf(state);printf(state_next);printf(d_state);printf(d_state_next);

    //fill device memory
    checkCudaErrors(hipMemcpy(d_state, state, mesh_width * mesh_width * mesh_height * sizeof(int), hipMemcpyHostToDevice));
    //printf("\nmemcpy happened fine\n");

    /* CUDA KERNEL CALL */
    int thread1D = 8; //MAX 64 for this GTX 1660Ti
    dim3 block(thread1D, thread1D, thread1D);
    dim3 grid(mesh_width / block.x, mesh_width / block.y, mesh_height / block.z);
    simple_vbo_kernel <<< grid, block >>> (dptr, mesh_width, mesh_width, mesh_height, g_fAnim, d_state, d_state_next);

    //next input state updated 
    checkCudaErrors(hipMemcpy(state, d_state_next, mesh_width * mesh_width * mesh_height * sizeof(int), hipMemcpyDeviceToHost));

    //int mid = mesh_width/2, x = mid, y = mid, z = mid, loc = (mesh_width * (y * mesh_width + x)) + z;
    //state[loc] = 1;
    //printf("\nvalue at state[mid] on host after cudaCpy = %d", state[loc]);

    //flush device memory
    checkCudaErrors(hipFree(d_state));checkCudaErrors(hipFree(d_state_next));

    //state_next freeing causing exceptions, TODO investigate reason
    //free(state_next); 

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void* data, unsigned int bytes, const char* filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE* fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char** argv, char* ref_file)
{
    char* reference_file = NULL;
    void* imageData = malloc(mesh_width * mesh_height * sizeof(float));

    // execute the kernel
    //launch_kernel((float4*)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width * mesh_height * sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width * mesh_height * sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
            mesh_width * mesh_height * sizeof(float),
            MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res,
    unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_loc_vbo_resource, state, state_next, d_state, d_state_next);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, loc_vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(0.0, 1.0, 0.4);
    glPointSize(1.0f);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_width * mesh_height); //added third multiplier 
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
        
    }
    if (loc_vbo) {
        deleteVBO(&loc_vbo, cuda_loc_vbo_resource);
    }
    free(state);free(state_next);
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
    case (27):

        glutDestroyWindow(glutGetWindow());
        return;

    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1 << button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char** argv, const GLuint& vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float* data = (float*)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char**)argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
            cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}
